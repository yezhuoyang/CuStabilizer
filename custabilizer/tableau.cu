#include <cstdio> 
#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>



__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}


__global__  void vector_add(float *A,float *B, float *C, int N) {
     int i=blockDim.x*blockIdx.x+threadIdx.x;
     if(i<N){
        C[i]=A[i]+B[i];
     }
}




int main() {


  int N=100;
  size_t size=N* sizeof(float);

  //Allocate vectors in host memory
  float* h_A =(float*) malloc(size);
  float* h_B =(float*) malloc(size); 
  float* h_C =(float*) malloc(size);

  //Initialize input vector
  for(int i=0;i<N;++i){
      h_A[i]=i;
      h_B[i]=i;
      h_C[i]=i;
  }

  float* d_A;
  hipMalloc(&d_A,size);
  float* d_B;
  hipMalloc(&d_B,size);
  float* d_C;
  hipMalloc(&d_C,size);    

  hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);  
  hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);
  hipMemcpy(d_C,h_C,size,hipMemcpyHostToDevice);
    

  int threadsPerBlock = 256;
  int blocksPerGrid = (N+threadsPerBlock-1)/threadsPerBlock;

  vector_add<<<blocksPerGrid,threadsPerBlock>>>(d_A,d_B,d_C,N);

 //copy result from device memory to host memory
 //h_C contains the result in host memory
 
 hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);

 hipFree(d_A);
 hipFree(d_B);
 hipFree(d_C);

  //Initialize input vector
 for(int i=0;i<N;++i){
      std::cout<<h_C[i]<<std::endl;   
  }
 
 return 0;
}