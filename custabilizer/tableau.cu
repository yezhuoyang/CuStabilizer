#include <cstdio> 
#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>
#include <string>
#include <bitset>
#include <sstream>
#include <fstream>


#define _CONTROLNOT 0
#define _HADAMARD 1
#define _PHASE 2
#define _PAULIX 3
#define _PAULIY 4
#define _PAULIZ 5
#define _CONTROLZ 6



struct Instruction {
    int type;
    int target;
    int control;

    Instruction(const int& tp,const int& contr,const int& targ):type(tp),target(targ),control(contr){

    }

};


// Overload the << operator outside the struct without friend
std::ostream& operator<<(std::ostream& os, const  Instruction& inst);



void  read_instructions_from_file(std::vector<Instruction>* instructionSet,const std::string &  filepath){
    std::ifstream file(filepath);
    std::vector<std::string> words;
    std::string word;
    if(!file.is_open()){
        std::cerr<<"File not exists!"<<std::endl;
    }
    std::string line;
    size_t count;
    int control;
    int target;
    Instruction* instpointer;
    while(getline(file,line)){
        words.clear();
        count=0;
        std::istringstream iss(line);
        while(iss>>word){
            words.push_back(word);
            count++;
        }
        if(count==2){
            if(words[0]=="h"){
                    target  = std::atoi(words[1].c_str());
                    instpointer=new Instruction(_HADAMARD,-1,target);
                    instructionSet->push_back(*instpointer);
            }
            else if(words[0]=="p"){
                    target  = std::atoi(words[1].c_str());
                    instpointer=new Instruction(_PHASE,-1,target);
                    instructionSet->push_back(*instpointer);
            }
            else if(words[0]=="x"){
                    target  = std::atoi(words[1].c_str());
                    instpointer=new Instruction(_PAULIX,-1,target);
                    instructionSet->push_back(*instpointer);
            }
            else if(words[0]=="y"){
                    target  = std::atoi(words[1].c_str());
                    instpointer=new Instruction(_PAULIY,-1,target);
                    instructionSet->push_back(*instpointer);
            }
            else if(words[0]=="z"){
                    target  = std::atoi(words[1].c_str());
                    instpointer=new Instruction(_PAULIZ,-1,target);
                    instructionSet->push_back(*instpointer);
            }
        }
        else if(count==3){
            if(words[0]=="c"){
                    control = std::atoi(words[1].c_str());
                    target  = std::atoi(words[2].c_str());
                    instpointer=new Instruction(_CONTROLNOT,control,target);
                    instructionSet->push_back(*instpointer);
            }
            else if(words[0]=="cz"){
                    control = std::atoi(words[1].c_str());
                    target  = std::atoi(words[2].c_str());
                    instpointer=new Instruction(_CONTROLZ,control,target);
                    instructionSet->push_back(*instpointer);
            }                    
        }
    }
    file.close();
}


__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}


__global__  void vector_add(float *A,float *B, float *C, int N) {
     int i=blockDim.x*blockIdx.x+threadIdx.x;
     if(i<N){
        C[i]=A[i]+B[i];
     }
}


#define getTableauElement(tableauMatrix,rowsize, row, col) (tableauMatrix[row*rowsize+col/8]&(0b10000000>>(col%8)))>>(7-(col%8))


#define setTableauValue(tableauMatrix,rowsize, row, col,value)       \
    do {                             \
        tableauMatrix[row*rowsize+col/8]=(value==1?(tableauMatrix[row*rowsize+col/8]|(0b00000000^(1<<(7-(col%8))))):(tableauMatrix[row*rowsize+col/8]&(0b11111111^(1<<(7-(col%8)))))); \
    } while (0)







__global__ void _P_cuda(unsigned char* tableauMatrix,size_t target,int qubit_num,int rowsize,int N){
    int row=threadIdx.x;
    if(row<N){
        int r=getTableauElement(tableauMatrix,rowsize, row, (2*qubit_num));
        int zi=getTableauElement(tableauMatrix,rowsize, row, target);
        int xi=getTableauElement(tableauMatrix,rowsize, row, (target+qubit_num));
        setTableauValue(tableauMatrix,rowsize, row, (2*qubit_num),((r+zi*xi)%2));
        setTableauValue(tableauMatrix,rowsize, row, target,((zi+xi)%2));
    }
}

__global__ void _H_cuda(unsigned char* tableauMatrix,size_t target,int qubit_num,int rowsize,int N){
    int row=threadIdx.x;
    if(row<N){
        int tmp=getTableauElement(tableauMatrix,rowsize, row, target);
        int z=getTableauElement(tableauMatrix,rowsize, row, (target+qubit_num));
        int r=getTableauElement(tableauMatrix,rowsize, row, (2*qubit_num));
        setTableauValue(tableauMatrix,rowsize, row, (2*qubit_num),((r+tmp*z)%2));
        setTableauValue(tableauMatrix,rowsize, row, target,z);
        setTableauValue(tableauMatrix,rowsize, row, (target+qubit_num),tmp);
    }
}


__global__ void _CNOT_cuda(unsigned char* tableauMatrix,size_t control,size_t target,int qubit_num,int rowsize,int N){
     int row=threadIdx.x;
    if(row<N){
        int zi=getTableauElement(tableauMatrix,rowsize, row, control);
        int xi=getTableauElement(tableauMatrix,rowsize, row, (control+qubit_num));
        int zj=getTableauElement(tableauMatrix,rowsize, row, target);
        int xj=getTableauElement(tableauMatrix,rowsize, row, (target+qubit_num));
        int r=getTableauElement(tableauMatrix,rowsize, row, (2*qubit_num));

        int multi=(zj*xi);
        int xorsum=(xj+zi+1)%2;

        setTableauValue(tableauMatrix,rowsize, row, (2*qubit_num),((r+multi+xorsum)%2));
        setTableauValue(tableauMatrix,rowsize, row, (target+qubit_num),((xi+xj)%2));
        setTableauValue(tableauMatrix,rowsize, row, control,((zi+zj)%2));
    }   
}


void P_cuda(int blocksPerGrid,int threadsPerBlock,unsigned char* tableauMatrix,size_t target,int qubit_num,int rowsize,int N){
    _P_cuda<<<blocksPerGrid,threadsPerBlock>>>( tableauMatrix,target,qubit_num,rowsize,N);
}

void H_cuda(int blocksPerGrid,int threadsPerBlock,unsigned char* tableauMatrix,size_t target,int qubit_num,int rowsize,int N){
    _H_cuda<<<blocksPerGrid,threadsPerBlock>>>( tableauMatrix,target,qubit_num,rowsize,N);
}

void CNOT_cuda(int blocksPerGrid,int threadsPerBlock,unsigned char* tableauMatrix,size_t control,size_t target,int qubit_num,int rowsize,int N){
    _CNOT_cuda<<<blocksPerGrid,threadsPerBlock>>>( tableauMatrix,control,target,qubit_num,rowsize,N);
}



// Function to print the binary representation of a char
void printBinary(char ch) {
    for (int i = 7; i >= 0; --i) { // Loop from 7 to 0 to get bits from MSB to LSB
        std::cout << ((ch >> i) & 1);
    }
}




//How to get element tableauMatrix[i][j]?
//  The byte contain the bit infor is  tableauMatrix[i*rowsize+j/8]
//  The index of the element in this byte is: j-8*(j/8)


//How to get the exact bit value of index k from a char A?
// int bit = (A >> k) & 1;


//If I have two bytes A and B, I want to get the XOR of the bit of A,B with index k?
//  ((A ^ B)>>k)&1;


//If I have two bytes A and B, I want to get the multplication of the bit of A,B with index k?
//  ((A & B)>>k)&1;


/*
int main() {


  int num_qubit=5;

  int threadNum=2*num_qubit;


  // Every row is processed in a single thread, every thread is exactly one row of the tableau
  // Every tableau is process in a block, every block is exactly processed in one block
  int rowsize=((2*num_qubit+1)+7)/8; 
  int size=rowsize*(2*num_qubit);

  unsigned char* tableauMatrix =(unsigned char*) malloc(size);

  //Initialize the tableau
  for(int i=0;i<size;++i){
      tableauMatrix[i]=0;
  }

  int tmpindex;
  for(int k=0;k<2*num_qubit;k++){
       setTableauValue(tableauMatrix,rowsize, k, k,1);
  }


  unsigned char* cutableauMatrix;
  cudaMalloc(&cutableauMatrix,size);
  //checkCudaError("cudaMalloc");

  cudaMemcpy(cutableauMatrix,tableauMatrix,size,cudaMemcpyHostToDevice); 
  //checkCudaError("cudaMemcpy to device");  

  int threadsPerBlock = 2*num_qubit;
  int blocksPerGrid =1;

  _H_cuda<<<blocksPerGrid, threadsPerBlock>>>(cutableauMatrix,1,num_qubit,rowsize,2*num_qubit);
  _CNOT_cuda<<<blocksPerGrid, threadsPerBlock>>>(cutableauMatrix,0,1,num_qubit,rowsize,2*num_qubit);



  //checkCudaError("Kernel launch");
  cudaDeviceSynchronize();
  //checkCudaError("Kernel execution");

  cudaMemcpy(tableauMatrix,cutableauMatrix,size,cudaMemcpyDeviceToHost); 
  //checkCudaError("cudaMemcpy to host");

  cudaFree(cutableauMatrix);
  //checkCudaError("cudaFree");

  //show_tableau_bit(tableauMatrix,num_qubit);  
  //show_tableau_char(tableauMatrix,num_qubit);  

  return 0;
}
*/